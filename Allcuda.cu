#include "hip/hip_runtime.h"
/* Programme of Viscous fingering */
/* applicable to the pure viscous fingering.							*/

/*To compile, type:
  gcc -O3 -o a.out ***.c -lm -lfftw -lrfftw
  gcc -O3 -o a.out ***.c -lm -lfftw -lrfftw
  To run the programme type "./a.out data rout"

  This produce a family of output files "rout_Suffix" which
  contain the various informations on the run.

  For cluster

  icc -O3 -tpp7 -xP -rcd -o a.out ****.c -I./myfftw/include ./myfftw/lib/librfftw.a ./myfftw/lib/libfftw.a
/*programme for postive and negtive R  */

#include <stdlib.h>
#include </usr/local/cuda/include/hipfft/hipfftw.h>
#include </usr/local/cuda/include/hipfft/hipfft.h>

#include </usr/local/cuda/include/cuda_device_runtime_api.h>

#include <stdio.h>
//#include <hipfft/hipfftw.h>
//#include <rfftw.h>
#include <math.h>
#include <string.h>
#include "/usr/local/cuda/include/hip/hip_runtime.h"
#include "/usr/local/cuda/include/"
#define PURE_DF  1

#define CELX 8192  		/* Number of grids or meshs according to x */
#define CELY 128       /* Number of grids or meshs according to y */
#define Lx   32768     /* length of the H-S cell */
#define Ly   512		/* width of the H-S cell equal to the Pe */

#define M_PI_2	1.57079632679489661923
#define M_2PI	6.2831853071795864769

#define tfinv hipfftExecC2R
#define tf hipfftExecR2C
#define ALLOC(var,TYPE,num_objects,routine) \
	var = (TYPE *) calloc((size_t)num_objects,sizeof(TYPE)); \
	if (var == NULL){\
		puts("Error in <"#routine">: Allocation of "#var" failed.");\
		puts("Exiting.");\
		exit(EXIT_FAILURE);\
	}
#define cudaALLOC(var,TYPE,num_objects,routine) \
	hipMalloc((void**)&var, sizeof(TYPE)*num_objects);  \
	if (var == NULL){\
		puts("Error in GPU <"#routine">: Allocation of "#var" failed.");\
		puts("Exiting.");\
		exit(EXIT_FAILURE);\
	}

typedef hipfftReal real;			/* Define type 'real', precision defined in <fftw.h> */
typedef hipfftComplex complexy;   /* Define type complex using type real */
hipfftHandle pfft,pinv;

extern int rand();			/* extern : permet de dÃ©clarer une fonction dÃ©finie ailleurs */
double arand();
void definitions(real *,real *,real *,real *,real *,real *);
void condition_initiale(int choix);
void dK_x(complexy *,complexy *);
void dK_y(complexy *,complexy *);
void lap();
void integration();
void read_parameters(char *file);
void premiere_fois();
void tfi(complexy *, real *);
void pmoyen(char *filein);
void mean_wavenumber(char *filein);
void write_data(char *filein);
/*void detecteur();*/

real *c,*Psi;
real *cG,*PsiG; 
real *kx,*ky,*k2,*k2t;
real *kxG,*kyG,*k2G,*k2tG;
real *dtk2,*dt2k2;
real *dtk2G,*dt2k2G;
complexy *cc,*PP,*ccold,*JJold;
complexy *ccG,*PPG,*ccoldG,*JJoldG;

int    option,nt2,nt3,alea;
int    compteur = 0;
int    compteurb = 0;

int    back,front;

double tprofil,tconc,tld,total_time,integration_time;
double taille,dt2,R,dt,eps,width;
/***********************************************************************/


__global__ void definitions1(real* Kx, double interm, int CELX)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < CELX/2+1)
		Kx[idx] = idx*interm;	
}


__global__ void definitions2(real* Ky, double interm)
{
	int idy = blockIdx.x * blockDim.x + threadIdx.x;
	Ky[idy] = (((idy+CELY/2-1)%CELY)-CELY/2+1)*interm;	
}

__global__ void definitions3(int idint, real* K2, real* K2T, real* Dtk2,real* Dt2k2, real* Kx, real* Ky, double eps,double dt, double dt2, int CELX)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int id = idx+idint;
	if(idx < CELX/2+1)
	{
		K2[id] = eps*Ky[idy]*Ky[idy]+Kx[idx]*Kx[idx];
		K2T[id] = Ky[idy]*Ky[idy]+Kx[idx]*Kx[idx];
		Dtk2[id] = exp(-dt*K2[id]);
		Dt2k2[id] = dt2*K2[id];
	}
}


__global__ void initialize_zero(real* Psi)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	Psi[id] = 0;

}	

__global__ void dk_xloop(int idint, complexy* dv, complexy *x, int CELX)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < CELX/2 + 1)
	{
		int id =  idx + idint;
		dv[id].x = -kx[idx]*x[id].y;
		dv[id].y = kx[idx]*x[id].x;
	}

}

__global__ void dk_yloop(int idint, complexy* a, complexy *b, int idy, int CELX)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < CELX/2 + 1)
	{
		int id = idx + idint;
		dv[id].x = -ky[idy]*x[id].y;
		dv[id].y = ky[idy]*x[id].x;
	}

}


__global__ void tfiloop(int x)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	x[id] = x[id]/taille;
}


__global__ void premeirefois1(real *Jold, real *N, real *cx, real *cy, real *Psix, real *Psiy)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	Jold[id] = Psiy[id]*cx[id]-Psix[id]*cy[id];
	N[id] = Psix[id]*cx[id]+Psiy[id]*cy[id];
}


__global__ void premierefois2(complexy *cc,complexy *ccold,complexy *JJold, double dt, real *k2, int CELY, int CELX)
{
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id < CELY*(CELX/2+1))
	{
		ccold[id].x = cc[id].x;
		ccold[id].y = cc[id].y;
		cc[id].x = -dt*JJold[id].x+(1.-k2[id]*dt)*cc[id].x;
		cc[id].y = -dt*JJold[id].y+(1.-k2[id]*dt)*cc[id].y;		
	}
}
__global void premierefois3(complexy *PP, double R, complexy *NN, complexy *ccy, real *k2t, int CELX, int CELY)
{
	if(id == 0)
	{
		PP[id].x = 0;
		PP[id].y = 0;
	}
	else if(id < CELY * (CELX/2+1))
	{
		PP[id].x = R*(NN[id].x+ccy[id].x)/k2t[id];
		PP[id].y = R*(NN[id].y+ccy[id].y)/k2t[id];
	}
}

__global__ void integration1(real *J, real *N, real *Psiy, real *Psix,real *cx, real *cy)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	J[id] = Psiy[id]*cx[id]-Psix[id]*cy[id];
	N[id] = Psix[id]*cx[id]+Psiy[id]*cy[id];

}

__global__ void integration2(complexy *ccnew,complexy *cc,double dt, complexy *JJ, complexy *JJold, real *dtk2, int CELX, int CELY)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if(id < CELY*(CELX/2+1))
	{
		ccnew[id].x = cc[id].x-dt*(1.5*JJ[id].x-0.5*JJold[id].x);
		ccnew[id].x = ccnew[id].x*dtk2[id];
		ccnew[id].y = cc[id].y-dt*(1.5*JJ[id].y-0.5*JJold[id].y);
		ccnew[id].y = ccnew[id].y*dtk2[id];
	}
}

__global__ void integration3(complexy *PP, complexy *NN, complexy *ccy, real *k2t, double R, int CELX, int CELY)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if(id == 0)
	{
		PP[id].x = 0.;
		PP[id].y = 0.;
	}
	else if(id < CELY*(CELX/2+1))
	{
		PP[id].x = R*(NN[id].x+ccy[id].x)/k2t[id];
		PP[id].y = R*(NN[id].y+ccy[id].y)/k2t[id];
	}
}


__global__ void integration4(real *Jnew, real *Psix, real *Psiy, real *cx, real *cy)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	Jnew[id] = Psiy[id]*cx[id]-Psix[id]*cy[id];

}

__global__ void integration5(complexy *cc, complexy *ccold, double dt2, complexy *JJnew, complexy *JJ, real *dt2k2, complexy *cc)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if(id < CELY*(CELX/2+1))
	{
		ccold[id].x = cc[id].x;
		ccold[id].y = cc[id].y;
		cc[id].x = cc[id].x-dt2*(JJnew[id].x+JJ[id].x)-dt2k2[id]*(ccnew[id].x+cc[id].x);
		cc[id].y = cc[id].y-dt2*(JJnew[id].y+JJ[id].y)-dt2k2[id]*(ccnew[id].y+cc[id].y);
		JJold[id].x = JJ[id].x;
		JJold[id].y = JJ[id].y;
	}
}



int main(int argc,char **argv)
{
	int seed;
        int i,idx,idy,id,idint,npas;
	int size[2],nligne,opt;
	int *sizeG;
	//cudaALLOC(sizeG,int,2,main);

	double interm;
	char sim_name[60];

	strcpy(sim_name,argv[2]);
								/* strcpy(dest,source) copie la chaine source dans dest							 */
								/* NB: dest doit etre assez grand pour contenir source + un caractere nul de fin */
								/*     donc on a: sim_name = "rout "											 */

	/*******  DEFINITIONS  **********************************/
		/*hipMalloc((void**)&cc, sizeof(hipfftComplex)*CELX*CELY);
		if(cc==NULL)
			printf("Gotcha\n");*/
        cudaALLOC(c,real,CELX*CELY,main);

        cudaALLOC(Psi,real,CELX*CELY,main);
        cudaALLOC(kx,real,CELX/2+1,main);
        cudaALLOC(ky,real,CELY,main);
        cudaALLOC(k2,real,CELY*(CELX/2+1),main);
        cudaALLOC(k2t,real,CELY*(CELX/2+1),main);
        cudaALLOC(dtk2,real,CELY*(CELX/2+1),main);
        cudaALLOC(dt2k2,real,CELY*(CELX/2+1),main);

        cudaALLOC(cc,complexy,CELY*(CELX/2+1),main);
        cudaALLOC(PP,complexy,CELY*(CELX/2+1),main);
        cudaALLOC(ccold,complexy,CELY*(CELX/2+1),main);
        cudaALLOC(JJold,complexy,CELY*(CELX/2+1),main);

        //cudaALLOC(cG,real,CELX*CELY,main);
        //cudaALLOC(PsiG,real,CELX*CELY,main);
        //cudaALLOC(kxG,real,CELX/2+1,main);
        //cudaALLOC(kyG,real,CELY,main);
        //cudaALLOC(k2G,real,CELY*(CELX/2+1),main);
        //cudaALLOC(k2tG,real,CELY*(CELX/2+1),main);
        //cudaALLOC(dtk2G,real,CELY*(CELX/2+1),main);
        //cudaALLOC(dt2k2G,real,CELY*(CELX/2+1),main);
 
        //cudaALLOC(ccG,complexy,CELY*(CELX/2+1),main);
        //cudaALLOC(PPG,complexy,CELY*(CELX/2+1),main);
        //cudaALLOC(ccoldG,complexy,CELY*(CELX/2+1),main);
        //cudaALLOC(JJoldG,complexy,CELY*(CELX/2+1),main);

	size[0]=CELY;
	size[1]=CELX;
	hipfftHandle *ptr;
    ptr = &pfft;
    hipfftHandle *ptr_inv;
    ptr_inv = &pinv;
    

	hipfftPlan2d(&pfft, size[0], size[1], HIPFFT_R2C);
	hipfftPlan2d(ptr_inv, size[0], size[1], HIPFFT_C2R);
	
	
	taille = (double)CELX*CELY;
	read_parameters(argv[1]);	// argv[1] a Ã©tÃ© remplacÃ© par "condinit" 

	npas = integration_time/dt+1;
	nligne = npas/(integration_time/tld);
	front = 10.;
	back = 0.;

/*	seed = 1000;
	srand(seed);*/
	/* initialize random number generator */

	write_data(sim_name);
	definitions(kx,ky,k2,k2t,dtk2,dt2k2);
	printf("****EVERYTHING FINE TILL LINE 134****\n");
	condition_initiale(option);
	/*for(i=1;i<CELX*CELY;i++)
	{
		printf("haha = %f\n",c[i]);
	}*/
	printf("ARAND = %f\n",arand());
	pmoyen(sim_name);
	mean_wavenumber(sim_name);
	premiere_fois();

	for (i = 1; i <= npas; i++){
		printf("Hi\n");
		  total_time = total_time+dt;
		  integration();

	  if(((i+1)%nligne)==0){
	    if(front-back<0){
	    i = npas+1;
	    printf("front=%f back=%f\n",front,back);
	    printf("Les deux fronts se rejoignent a t=%f\n",total_time);
	    }
	    
	    tfi(cc,c);
	    tfi(PP,Psi);

	    pmoyen(sim_name);
	    mean_wavenumber(sim_name);
	   
		tf(pfft,c,cc);

		tf(pfft,Psi,PP);		
	  }
	}
// 	printf("Temps final de la simulation = %f\n",total_time);
// 	printf("------------------------------------------------------------\n\n");

	hipFree(c);
	hipFree(ccold);
	hipFree(JJold);
	hipFree(cc);
	hipFree(Psi);
	hipFree(PP);
	hipFree(ky);
	hipFree(kx);
	hipFree(k2);
	hipFree(k2t);
	hipFree(dtk2);
	hipFree(dt2k2);
	hipFree(cG);
	hipFree(ccoldG);
	hipFree(JJoldG);
	hipFree(ccG);
	hipFree(PsiG);
	hipFree(PPG);
	hipFree(kyG);
	hipFree(kxG);
	hipFree(k2G);
	hipFree(k2tG);
	hipFree(dtk2G);
	hipFree(dt2k2G);
	hipFree(pfft);
	hipFree(pinv);
	return 0;
}
/*&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&*/
/*void detecteur()
  {	int  idx,idy,id,idint;
  real *cmx;
  double ddx;

  ddx = (double)Lx/CELX;
  for (idx=0; idx<CELX ; ++idx) cmx[idx] = 0.;
  for (idy = 0; idy < CELY; ++idy){
  idint = idy*CELX;
  for (idx=0 ; idx<CELX ; ++idx){
  id = idint+idx;
  cmx[idx] = cmx[idx]+c[id];
  }
  }

  for(idx=0;idx<CELX;idx++)
  cmx[idx]=cmx[idx]/CELY;


  }*/
/*&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&*/
void pmoyen(char *filein) // filein = rout
{
	int  idx,idy,id,idint;
	double interm,moyenne1,moyenne2,pos,tmp;
	double beginning,area,M1,M2,M3,Mtot,temp,sfing;
	real *cmx,*cmxG;
	double ddx,ddy;
	int d1,d2,c99;
	int longueur=0;
	char file_name[60];
	FILE *fp;
	real *ccc;
	ALLOC(ccc,real,CELX*CELY,pmoyen);
	ALLOC(cmx,real,CELX,pmoyen);
	//cudaALLOC(cmxG,real,CELX,pmoyen);
	hipMemcpy(ccc,c,sizeof(real)*CELX*CELY,hipMemcpyDeviceToHost);

	while (filein[longueur] != '\0') longueur += 1;
	longueur += 2;
	M1 = 0.;
	M2 = 0.;
	M3 = 0.;
	Mtot = 0.;
	printf("longueur=%d\n",longueur);
	/*-------------------  Mean profile versus x   ---------------------*/

	ddx = (double)Lx/CELX;
	ddy = (double)Ly/CELY;
	printf("ddx = %f\n",ddx);
	printf("ddy = %f\n",ddy);

	for (idx=0; idx<CELX ; ++idx) cmx[idx] = 0.;


	for (idy = 0; idy < CELY; ++idy){
		idint = idy*CELX;
		for (idx=0 ; idx<CELX ; ++idx){
			id = idint+idx;
			cmx[idx] = cmx[idx]+ccc[id];
		}
	}
	/*----------------   Mixing length ------------------------*/
	moyenne2 = cmx[0]/CELY;
	for(idx=0;idx<CELX;idx++)
		cmx[idx]=cmx[idx]/CELY;

	for(idx=0;idx<CELX;idx++){
		moyenne1 = moyenne2;
		moyenne2 = cmx[idx];
		if(moyenne2>0.01000 && moyenne1<0.01000) back=idx;
		if(moyenne2<0.01000 && moyenne1>0.01000) front=idx;
	}

	/*------   First moment (mean position of the gravity center) -----------*/

	for(idx=0;idx<CELX-1;idx+=2){
		M1 += 2*cmx[idx]*idx;
		Mtot += 2*cmx[idx];
		M1 += 4*cmx[idx+1]*(idx+1);
		Mtot += 4*cmx[idx+1];
	}

	Mtot = Mtot*ddx/3.;
	M1 = M1*ddx*ddx/Mtot/3.;
	printf("Mtot = %f\n",Mtot);


	/*----------------   Second and third moments -------------*/
	for(idx=0;idx<CELX-1;idx+=2){
		interm = idx*ddx-M1;
		temp = interm*interm;
		M2 += 2*cmx[idx]*temp;
		M3 += 2*cmx[idx]*temp*interm;
		interm = (idx+1)*ddx-M1;
		temp = interm*interm;
		M2 += 4*cmx[idx+1]*temp;
		M3 += 4*cmx[idx+1]*temp*interm;
	}
	M2 = M2*ddx/Mtot/3.;
	M3 = M3*ddx/Mtot/3.;
	sfing = M2-(width*width)/12.-2.*total_time;
	sfing = sqrt(sfing);
	printf("M3 = %f\n",M3);
	/*----------------Writing in files -------------*/


	strcpy(file_name,filein);
	strcat(file_name,"L");
	fp = fopen(file_name,"a");
	fprintf(fp,"%f %f\n",total_time,(front-back)*ddx);
	fclose(fp);



	strcpy(file_name,filein);
	strcat(file_name,"M1");
	fp = fopen(file_name,"a");
	fprintf(fp,"%f %f %f\n",total_time,M1,Mtot);
	fclose(fp);

	strcpy(file_name,filein);
	strcat(file_name,"M2");
	fp = fopen(file_name,"a");
	fprintf(fp,"%f %f\n",total_time,M2);
	fclose(fp);

	strcpy(file_name,filein);
	strcat(file_name,"M3");
	fp = fopen(file_name,"a");
	fprintf(fp,"%f %f\n",total_time,M3);
	fclose(fp);

	strcpy(file_name,filein);
	strcat(file_name,"SF");
	fp = fopen(file_name,"a");
	fprintf(fp,"%f %f\n",total_time,sfing);
	fclose(fp);

	if((compteur%nt2)==0){           /* Printing every tprofil times */


		/*------ -------------- Mean profile along X ------------*/

		strcpy(file_name,filein);
		strcat(file_name,"X");
		fp = fopen(file_name,"a");
		for(idx=0;idx<CELX;idx++){
			interm = idx*ddx;
			moyenne2 = cmx[idx];
			fprintf(fp,"%f %f %d\n",interm,moyenne2,idx);
		}
		fprintf(fp,"\n");
		fclose(fp);

	}
	/*-------------------- Concentration matrix------------*/
	if((compteur%nt3)==0){
		d1 = fmod(compteurb,10);
		d2 = fmod((compteurb-d1)/10,10);
		strcpy(file_name,filein);
		strcat(file_name,"_c");

		file_name[longueur]=d2+'0';
		file_name[longueur+1]=d1+'0';
		file_name[longueur+2]='\0';
		fp = fopen(file_name,"w");
		for (idx=0 ; idx<CELX ; ++idx){
			for (idy = 0; idy < CELY; ++idy){
				idint = idy*CELX;
				id = idx+idint;
				fprintf(fp,"%f ",ccc[id]);
			}
			fprintf(fp,"\n");

		}
		fclose(fp);


		/*-------------------- Stream function matrix   ------------*/

		/*	  strcpy(file_name,filein);
			  strcat(file_name,"_p");
			  file_name[longueur]=d2+'0';
			  file_name[longueur+1]=d1+'0';
			  file_name[longueur+2]='\0';

			  fp = fopen(file_name,"w");
			  for (idx=0 ; idx<CELX ; ++idx){
			  for (idy = 0; idy < CELY; ++idy){
			  idint = idy*CELX;
			  id = idx+idint;
			  fprintf(fp,"%f ",Psi[id]);
			  }
			  fprintf(fp,"\n");
			  }
			  fclose(fp);*/

		compteurb += 1;
	}

	compteur += 1;
	free(cmx);

}
/**************************************************************************/
void mean_wavenumber(char *filein)
{
	int  i,j,idx,idy,id,idint,imax;
	char file_name[60];
	real *ccc;
	ALLOC(ccc,real,CELX*CELY,pmoyen);
	hipMemcpy(ccc,c,sizeof(real)*CELX*CELY,hipMemcpyDeviceToHost);

	real *temp,*cmy,*tempG,*cmyG;
	double *P;
	complexy *ttemp,*ttempG;
	double ddy,Pmax,kmean,Ptot;
	int nmode;
	FILE *fp,*gp,*pp,*lp;

	ALLOC(temp,real,CELY*CELX,mean_wavenumber);
	ALLOC(cmy,real,CELY,mean_wavenumber);
	ALLOC(ttemp,complexy,CELY*(CELX/2+1),mean_wavenumber);
	cudaALLOC(tempG,real,CELY*CELX,mean_wavenumber);
	cudaALLOC(cmyG,real,CELY,mean_wavenumber);
	cudaALLOC(ttempG,complexy,CELY*(CELX/2+1),mean_wavenumber);
	ALLOC(P,double,CELY/2+1,mean_wavenumber);

 	strcpy(file_name,filein);
	strcat(file_name,"K");
	fp = fopen(file_name,"a");
 	strcpy(file_name,filein);
	strcat(file_name,"Kmax");
	pp = fopen(file_name,"a");
 	strcpy(file_name,filein);

	lp = fopen(file_name,"a");

	/********* Initialization ***************/
	nmode = 0;
	kmean = 0.;
	Pmax = 0.;
	Ptot = 0.;

	for (idy = 0; idy < CELY; ++idy){
	  idint = idy*CELX;
	  cmy[idy] = 0.;
	  for (idx=0 ; idx<CELX ; ++idx){
	    id = idx+idint;
	    temp[id] = 0.;
	  }
	}

	/********* Mean Profile along Y ***************/

   	for (idy = 0; idy < CELY; ++idy){
		idint = idy*CELX;
		for (idx=0 ; idx<CELX ; ++idx){
			id = idx+idint;
			cmy[idy] = cmy[idy]+ccc[id];
			}
		}

   	for (idy = 0; idy < CELY; ++idy){
		idint = idy*CELX;
		cmy[idy] = cmy[idy]/CELX;
		for (idx=0 ; idx<CELX ; ++idx){
			id = idx+idint;
			temp[id] = cmy[idy];
			}
		}

	/********** Power spectrum ************/
	/*for(i=0;i<CELX*CELY;i++)
	{
		printf("%f\n",temp[i]);
		if(temp[i]!=0)
			break;
	}*/
		//TEMP HAS NON ZERO VALUES
	hipMemcpy(tempG, temp,CELY*CELX*sizeof(real), hipMemcpyHostToDevice);
	tf(pfft,tempG,ttempG);
	hipMemcpy(ttemp, ttempG, CELY*(CELX/2+1)*sizeof(complexy), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
/*	for(i=0;i<CELY*(CELX/2+1);i++)
	{

		printf("%f\n",temp[i]);
		if(ttemp[i].x!=0)
			printf("%f\n",ttemp[i].x);
			
	}
*/
	//printf("i=%d\n",i);
	//printf("CELY*(CELX/2+1)=%d\n",CELY*(CELX/2+1));

	

	for(idy = 1; idy <= CELY/2; ++idy){
		idint = idy*(CELX/2+1);
		for (idx=0 ; idx<CELX/2+1 ; idx++){
			id = idx+idint;
			P[idy] += ttemp[id].x*ttemp[id].x+ttemp[id].y*ttemp[id].y;
			  }
		P[idy] = sqrt(P[idy]);
		Ptot += P[idy];
		if(P[idy]>Pmax){
		  imax = idy;
		  Pmax = P[idy];
		}	
		nmode +=1;
	}
	/*for(i=0;i<CELY/2+1;i++)
		printf("P=%lf\n",P[i]);*/
	printf("ptot=%lf\n",Ptot);
	printf("pmax=%lf\n",Pmax);

	for(idy = 1; idy <= CELY/2; ++idy){
	  kmean += P[idy]*idy/Ptot;
	}
	printf("kmean=%f\n",kmean);

	fprintf(fp,"%f %f\n",total_time,kmean);
	fprintf(pp,"%f %d %f\n",total_time,imax,Pmax);

	fclose(fp);
	fclose(pp);

	/*************** Trace du profil moyen ********/
	if((compteur%nt2)==0){
	  strcpy(file_name,filein);
	  strcat(file_name,"Y");
	  gp = fopen(file_name,"a");
	  for (idy = 0; idy < CELY; ++idy){
		ddy = idy*Ly/CELY;
		fprintf(gp,"%f %f\n",ddy,cmy[idy]);
	  }
	fprintf(gp,"\n");
	fclose(gp);
	}

	free(temp);
	free(ttemp);
	free(cmy);
	free(P);
	hipFree(tempG);
	hipFree(ttempG);
	hipFree(cmyG);
}
/**************************************************************************/
void tfi(complexy *xx,real *x)
{
	// int id;
	// complexy *xxG;
	// real *xG;

	// cudaALLOC(xxG,complexy,CELY*(CELX/2+1),tfi);
	// cudaALLOC(xG,real,CELX*CELY,tfi);

	// hipMemcpy(xxG, xx, CELY*(CELX/2+1)*sizeof(complexy), hipMemcpyHostToDevice);	
	tfinv(pinv,xx,x);
	// hipMemcpy(x, xG,CELY*CELX*sizeof(real), hipMemcpyDeviceToHost);

	// hipDeviceSynchronize();
	tfiloop<<<1024,1024>>>(x);
	// for(id=0;id<CELY*CELX;id++) x[id] = x[id]/taille;
	// hipFree(xxG);
	// hipFree(xG);
}
/**************************************************************************/
void premiere_fois()
{
	int id,idy;
	real *Jold,*N,*JoldG, *NG;
	real *cx,*cy;
	real *Psix,*Psiy;
	complexy *deriv,*ccy,*NN, *NNG;

	cudaALLOC(Jold,real,CELX*CELY,premiere_fois);
	cudaALLOC(N,real,CELX*CELY,premiere_fois);
	cudaALLOC(JoldG,real,CELX*CELY,premiere_fois);
	cudaALLOC(NG,real,CELX*CELY,premiere_fois);
	cudaALLOC(cx,real,CELX*CELY,premiere_fois);
	cudaALLOC(cy,real,CELX*CELY,premiere_fois);
	cudaALLOC(Psix,real,CELX*CELY,premiere_fois);
	cudaALLOC(Psiy,real,CELX*CELY,premiere_fois);
	cudaALLOC(deriv,complexy,CELY*(CELX/2+1),premiere_fois);
	cudaALLOC(NN,complexy,CELY*(CELX/2+1),premiere_fois);

	tf(pfft,c,cc);
	// hipMemcpy(cc, ccG, CELY*(CELX/2+1)*sizeof(complexy), hipMemcpyDeviceToHost);
	// hipDeviceSynchronize();

	// hipMemcpy(PsiG, Psi,CELY*CELX*sizeof(real), hipMemcpyHostToDevice);
	tf(pfft,Psi,PP);
	// hipMemcpy(PP, PPG, CELY*(CELX/2+1)*sizeof(complexy), hipMemcpyDeviceToHost);
	// hipDeviceSynchronize();

	total_time = total_time+dt;

	/*********** DERIVEES DANS L'ESPACE DE FOURIER  **************/
	dK_y(cc,deriv);
	tfi(deriv,cy);
	dK_x(cccc,deriv);
	tfi(deriv,cx);
	dK_y(PP,deriv);
	tfi(deriv,Psiy);
	dK_x(PP,deriv);
	tfi(deriv,Psix);
	hipFree(deriv);

	/*********** NON LINEARITES DANS L'ESPACE REEL  **************/
	tfi(cc,c);

	premierefois1<<<1024,1024>>>(Jold, N, cx, cy, Psix, Psiy);

	// for (id = 0; id < CELX*CELY; ++id)
	// {
	// 	Jold[id] = Psiy[id]*cx[id]-Psix[id]*cy[id];
	// 	N[id] = Psix[id]*cx[id]+Psiy[id]*cy[id];
	// }

	tf(pfft,c,cc);
	
	tf(pfft,Jold,JJold);


	tf(pfft,N,NN);
	hipFree(Jold);
	hipFree(N);

	/*********** TERMES LINEAIRES DANS L'ESPACE DE FOURIER  *******/

	premierefois2<<<513,1024>>>(cc,ccold,JJold,dt,k2, CELY, CELX);

	cudaALLOC(ccy,complexy,CELY*(CELX/2+1),integration);
	dK_y(cc,ccy);

	premierefois3<<<513,1024>>>(PP, R, NN, ccy, k2t, CELX, CELY)

	
	hipFree(ccy);
	hipFree(cx);
	hipFree(cy);
	hipFree(Psix);
	hipFree(Psiy);
	hipFree(NN);
	hipFree(JoldG);
	hipFree(NG);
	hipFree(JJoldG);
	hipFree(NNG);

}
/**************************************************************************/
void integration()
{
	int id,idy;
	real *J,*N,*JG,*NG;
	real *Psix,*Psiy;
	real *cx,*cy,*Jnew,*cnew,*cnewG, *JnewG;
	complexy *deriv,*ccy,*JJnew,*JJnewG,*ccnew,*JJ,*NN,*JJG,*NNG,*ccnewG;

	cudaALLOC(J,real,CELX*CELY,integration);
	cudaALLOC(N,real,CELX*CELY,integration);
	cudaALLOC(cx,real,CELX*CELY,integration);
	cudaALLOC(cy,real,CELX*CELY,integration);
	cudaALLOC(Psix,real,CELX*CELY,integration);
	cudaALLOC(Psiy,real,CELX*CELY,integration);
	cudaALLOC(Jnew,real,CELX*CELY,integration);
	cudaALLOC(cnew,real,CELX*CELY,integration);
	//cudaALLOC(cnewG,real,CELX*CELY,integration);
	//cudaALLOC(JG,real,CELX*CELY,integration);
	//cudaALLOC(NG,real,CELX*CELY,integration);
	//cudaALLOC(JnewG,real,CELX*CELY,integration);

	cudaALLOC(deriv,complexy,CELY*(CELX/2+1),integration);
	cudaALLOC(ccnew,complexy,CELY*(CELX/2+1),integration);
	cudaALLOC(JJ,complexy,CELY*(CELX/2+1),integration);
	cudaALLOC(ccy,complexy,CELY*(CELX/2+1),integration);
	cudaALLOC(JJnew,complexy,CELY*(CELX/2+1),integration);
	cudaALLOC(NN,complexy,CELY*(CELX/2+1),integration);
	//cudaALLOC(JJG,complexy,CELY*(CELX/2+1),integration);
	//cudaALLOC(NNG,complexy,CELY*(CELX/2+1),integration);
	//cudaALLOC(ccnewG,complexy,CELY*(CELX/2+1),integration);
	//cudaALLOC(JJnewG,complexy,CELY*(CELX/2+1),integration);

	dK_y(cc,deriv);
	tfi(deriv,cy);
	dK_x(cc,deriv);
	tfi(deriv,cx);
	dK_y(PP,deriv);
	tfi(deriv,Psiy);
	dK_x(PP,deriv);
	tfi(deriv,Psix);

	tfi(cc,c);
	
	integration1<<<1024,1024>>>(J,N,Psix,Psix,cx,cy);
	/*======  Espace de Fourier ===============================*/

	tf(pfft,J,JJ);
	tf(pfft,N,NN);
	hipFree(J);
	hipFree(N);
	tf(pfft,c,cc);

	integration2<<<513,1024>>>(ccnew,cc,dt,JJ,JJold,dtk2,CELX,CELY);

	dK_y(ccnew,ccy);  /* necessaire car c a change en cours de route */

	integration3<<<513,1024>>>(PP, NN, ccy, k2t, R, CELX, CELY);
	tfi(ccy,cy);
	hipFree(ccy);
	dK_x(ccnew,deriv);
	tfi(deriv,cx);
	dK_y(PP,deriv);
	tfi(deriv,Psiy);
	dK_x(PP,deriv);
	tfi(deriv,Psix);

	tfi(ccnew,cnew);

	integration4<<<1024,1024>>> (Jnew, Psix, Psiy, cx, cy);

	tf(pfft,Jnew,JJnew);
	tf(pfft,cnew,ccnew);

	integration5<<<513,1024>>> (cc,ccold,dt2,JJnew,JJ,dt2k2,ccnew);

	hipFree(JJnew);
	hipFree(ccnew);
	hipFree(deriv);
	hipFree(JJ);
	hipFree(cx);
	hipFree(cy);
	hipFree(Psix);
	hipFree(Psiy);
	hipFree(cnew);
	hipFree(Jnew);
	hipFree(NN);
	hipFree(JG);
	hipFree(NG);
	hipFree(JnewG);
	hipFree(cnewG);
	hipFree(JJG);
	hipFree(NNG);
	hipFree(JJnewG);
	hipFree(ccnewG);
	
}
/*=========================================================================*/
void condition_initiale(int choix)
{
	real* ccc;
	ALLOC(ccc, real, CELX*CELY, condition_initiale);
	
	printf("****CONDITION INITIALE STARTS\n****");
	int i,idx,idy,id,idint,finalea;
	double pos,ddx,z1,z2,kgauche,vgauche,interm,zut1, fixc;
	int debut,fin;
	ddx = (double)Lx/CELX;
	if(choix == 1){
		printf("Condition initiale = stepfunction + bruit sur c=1/2 \n");

		initialize_zero<<<1024,1024>>>(Psi);

		if(R<0.)debut = 1*(Lx-width)/5./ddx;
		else debut=4*(Lx-width)/5./ddx;
		/*	  fin = (Lx+width)/2./ddx;*/
		fin = debut + width/ddx;

		printf("debut=%f fin=%f largeur=%f milieu=%f\n",debut*ddx,fin*ddx,(fin-debut)*ddx,(fin+debut)*ddx/2.);

		finalea = alea*CELY;
		printf("finalea=%d\n",finalea);
		for(idy=0; idy<finalea;++idy)
		{
			zut1 = arand();
			printf("zut = %f\n",zut1);
		} 

		//printf("%f\n",zut1);
		for (idy = 0; idy < CELY; ++idy)
		{
			idint = idy*CELX;
			
			for (idx=0 ; idx<debut ; ++idx)
			{
				id = idx+idint;
				ccc[id] = 0.;
			}
			
			for (idx=debut ; idx<debut+1 ; ++idx)
			{
				id = idx+idint;
				fixc = 1./2.*(1.+0.001*arand());
				ccc[id]=fixc;
			}
			
			for (idx=debut+1 ; idx<fin ; ++idx)
			{
				id = idx+idint;
				ccc[id] = 1.;
			}
		
			for (idx=fin ; idx<fin+1 ; ++idx)
			{
				id = idx+idint;
				ccc[id] = 1-fixc;
			}

			for (idx=fin+1 ; idx<CELX; ++idx)
			{
				id = idx+idint;
				ccc[id] = 0.;
			}
}
//printf("c=%lf\n",c[id]);

}
	hipMemcpy(c, ccc, sizeof(real)*CELX*CELY, hipMemcpyHostToDevice);
}

/*=========================================================================*/
void definitions(real Kx[],real Ky[],real K2[],real K2T[],real Dtk2[],real Dt2k2[])
{
	printf("***INSIDE DEFINITION FUNCTION****\n");
	int  idx,idy,idint,id;
	double interm;

	interm = M_2PI/Lx;
	definitions1<<<5,820>>>(Kx, interm, CELX);

	interm = M_2PI/Ly;
	definitions2<<<1,128>>>(Ky,interm);

	for(idy=0;idy<CELY;idy+=1){
		idint = idy*(CELX/2+1);
		definitions3<<<5,820>>>(idint, K2, K2T, Dtk2, Dt2k2, Kx, Ky, eps, dt, dt2, CELX);
	}

	printf("***DEFINITIONS ENDS****\n");
}
/*=========================================================================*/
void dK_x(complexy x[],complexy dv[])
{
	int idx,idy,id,idint;
	for (idy = 0; idy < CELY; ++idy){
		idint = idy*(CELX/2+1);
		dk_xloop<<<5,820>>>(idint,dv,x, CELX);
		
		// for(idx=0;idx<CELX/2+1;idx+=1){
		// 	id = idx+idint;
		// 	dv[id].x = -kx[idx]*x[id].y;
		// 	dv[id].y = kx[idx]*x[id].x;
		// }
	}
}
/*=========================================================================*/
void dK_y(complexy x[],complexy dv[])
{
	int idx,idy,id,idint;

	for (idy = 0; idy < CELY; ++idy){
		idint = idy*(CELX/2+1);
		dk_yloop<<<5,820>>>(idint,dv,x, idy, CELX);
		
		// for(idx=0;idx<CELX/2+1;idx+=1){
		// 	id = idx+idint;
		// 	dv[id].x = -ky[idy]*x[id].y;
		// 	dv[id].y = ky[idy]*x[id].x;
		// }
	}
}
/*=========================================================================*/
void lap(complexy x[],complexy dx[])
{
	int  id;

	for (id = 0; id < CELY*(CELX/2+1); ++id){
		dx[id].x = -k2t[id]*x[id].x;
		dx[id].y = -k2t[id]*x[id].y;
	}
}
/*=========================================================================*/
void read_parameters(char *file)
{
	FILE *rp;

	printf("\n----------------------------------------------------------\n\n");
#if PURE_DF == 1
	printf(" Pure viscous fingering - no chemical reaction\n");
#endif
	printf("\n----------------------------------------------------------\n\n\n");

	rp = fopen(file,"r");
	fscanf(rp,"option = %d\n",&option);
	fscanf(rp,"alea = %d\n",&alea);
	fscanf(rp,"R = %lf\n",&R);
	fscanf(rp,"eps = %lf\n",&eps);
	fscanf(rp,"dt = %lf\n",&dt);
	fscanf(rp,"integration_time = %lf\n",&integration_time);
	fscanf(rp,"total_time = %lf\n",&total_time);
	fscanf(rp,"tld = %lf\n",&tld);
	fscanf(rp,"tprofil = %lf\n",&tprofil);
	fscanf(rp,"tconc = %lf\n",&tconc);
	fscanf(rp,"width = %lf\n",&width);

	printf("****INPUT FILE PARAMETERS READING STARTS****\n");

	printf("option = %d\n",option);
	printf("alea = %d\n",alea);
	printf("R = %lf\n",R);
	printf("eps = %lf\n",eps);
	printf("dt = %lf\n",dt);
	printf("int_time = %lf\n",integration_time);
	printf("total_time = %lf\n",total_time);
	printf("tld = %lf\n",tld);
	printf("tprofil = %lf\n",tprofil);
	printf("tconc = %lf\n",tconc);
	printf("width = %lf\n",width);

	printf("****INPUT FILE PARAMETERS READING ENDS****\n");


	dt2 = dt/2.;
	nt2 = tprofil/tld;
	nt3 = tconc/tld;
	printf(" R=%f  eps=%lf  l=%f  \n\n",R,eps,width);		/*printf(" R=%f  eps=%lf  l=%f  Pe=%d\n\n",R,eps,width,(double)Lx/CELX);*/
	printf(" Pe=%d dx=%10.8f dy=%10.8f A=%d dt=%f\n",Ly,(double)Lx/CELX,(double)Ly/CELY,CELX/CELY,dt);
	printf("   (CELX=%d CELY=%d Lx=%d Ly=%d)\n",CELX,CELY,Lx,Ly);
	printf("    nmodes=%d\n",CELX*CELY);
	printf("    integration_time=%f\n",integration_time);
	printf("    sigma_init = %lf\n",width*width/12.);
	fclose(rp);
}
/**************************************************************************/
double arand()
{
	double z;
	z = (double) (1.0*rand()/RAND_MAX);
	return(z);
}
/*=========================================================================*/
void write_data(char *filein)
{
	char file_name[60];
	int nnpas,nnligne,nn;
	FILE *fp;

	strcpy(file_name,filein);
	strcat(file_name,"D");
	printf("*** OPEN ROUTD TO CHECK IF EVERYTHING IS PRINTED CORRECTLY****\n");
	fp = fopen(file_name,"w");

#if PURE_DF == 1
	fprintf(fp,"\n-----------------------------------------------\n\n");
	fprintf(fp,"  Pure viscous fingering - no chemical reaction\n\n");
#endif

	fprintf(fp,"  Donnees de la simulation %s\n\n",filein);
	fprintf(fp,"-----------------------------------------------\n");

	fprintf(fp,"Pe = %d          A=%f\n\n",Ly,(double)Lx/Ly);

	fprintf(fp,"CELX = %d\n",CELX);
	fprintf(fp,"CELY = %d\n",CELY);
	fprintf(fp,"Lx = %d\n",Lx);
	fprintf(fp,"Ly = %d\n",Ly);
	fprintf(fp,"dx = %4.2f\n",(double)Lx/CELX);
	fprintf(fp,"dy = %4.2f\n\n",(double)Ly/CELY);

	fprintf(fp,"CI = %d\n",option);
	fprintf(fp,"alea = %d\n",alea);
	fprintf(fp,"R = %lf\n",R);
	fprintf(fp,"eps = %lf\n",eps);
	fprintf(fp,"width = %lf\n",width);
	fprintf(fp,"dt = %lf\n",dt);
	fprintf(fp,"tld = %lf\n",tld);
	fprintf(fp,"tprofil = %lf\n",tprofil);
	fprintf(fp,"tconc = %lf\n",tconc);
	fprintf(fp,"total_time = %lf\n",total_time);
	fprintf(fp,"integ_time = %lf\n",integration_time);

	nnpas = integration_time/dt+1;
	nnligne = integration_time/tld;
	nn = integration_time/tprofil;
	fprintf(fp,"\nnpas=%d    nligne=%d  nprofil=%d\n",nnpas,nnligne,nn);

	nn = integration_time/tconc;
	fprintf(fp,"\nnconc=%d\n",nn);
	fprintf(fp,"-----------------------------------------------\n\n");

	fclose(fp);
}
